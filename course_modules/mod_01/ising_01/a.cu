
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>


#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
//#include "cudaDmy.cuh"

#ifdef __INTELLISENSE__
//#define __global__
#endif

//#include <cuda

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int cuda_main() {
    std::cout << "A" << std::endl;
    cuda_hello<<<1,1>>>(); 
    std::cout << "B" << std::endl;
    return 0;
}